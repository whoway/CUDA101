#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NSTREAM 4
#define BDIM 128

void initialData(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        for (int i = 0; i < 99999; ++i)
        {
            C[idx] = A[idx] + B[idx];
        }
    }
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 1 << 18;
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *gpuRef;
    hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault);
    hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(gpuRef,  0, nBytes);


    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // invoke kernel at host side
    dim3 block (BDIM);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x,
            block.y);

    // grid parallel operation
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
    {
        hipStreamCreate(&stream[i]);
    }

    hipEventRecord(start, 0);

    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]);
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset],
                &d_C[ioffset], iElem);
        hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
                              hipMemcpyDeviceToHost, stream[i]);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float execution_time;
    hipEventElapsedTime(&execution_time, start, stop);

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM,
           execution_time, (nBytes * 2e-6) / execution_time );

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(gpuRef);

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // destroy streams
    for (int i = 0; i < NSTREAM; ++i)
    {
        hipStreamDestroy(stream[i]);
    }

    hipDeviceReset();
    return(0);
}
