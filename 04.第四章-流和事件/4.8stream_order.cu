#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
    return;
}

__global__ void infiniteKernel()
{
    while(true)
    {
    }
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    int nElem = 32;

    // malloc host pinned memory
    float *pinned_A;
    size_t nBytes = nElem * sizeof(float);
    ErrorCheck(hipHostAlloc((void**)&pinned_A, nBytes, hipHostMallocDefault), __FILE__, __LINE__);

    initialData(pinned_A, nElem);
    // allocate gpu global memory
    float *d_A;
    hipMalloc((float**)&d_A, nBytes);

    //calculate on GPU
    dim3 block (nElem);
    dim3 grid  (2);
    infiniteKernel<<<grid, block>>>();

    // transfer data from host to device
    hipStream_t data_stream;
    //hipStreamCreate(&data_stream); // blocking stream
    hipStreamCreateWithFlags(&data_stream, hipStreamNonBlocking); //non-blocking stream

    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);
    hipEvent_t cp_evt;
    ErrorCheck(hipEventCreate(&cp_evt), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(cp_evt, data_stream),__FILE__, __LINE__);

    //wait for data copy to complete
    hipEventSynchronize(cp_evt);
    printf("Event cp_evt is finished\n");

    hipDeviceSynchronize();

    hipHostFree(pinned_A);
    hipFree(d_A);
    hipStreamDestroy(data_stream);
    hipEventDestroy(cp_evt);
    hipDeviceReset();

    return 0;
}
