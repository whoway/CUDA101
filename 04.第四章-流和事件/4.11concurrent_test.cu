#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if(deviceProp.concurrentKernels)
    {
        printf("cocurrent kernel is supported on this GPU, begin to execute kernel_1\n");

    }
    else
    {
        printf("cocurrent kernel is not supported on this GPU\n");
    }

    hipDeviceReset();
    return 0;
}
