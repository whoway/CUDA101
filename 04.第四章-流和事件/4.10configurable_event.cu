#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

__global__ void infiniteKernel()
{
    while(true)
    {
    }
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    int nElem = 32;

    // malloc host pinned memory

    //calculate on GPU
    dim3 block (nElem);
    dim3 grid  (1);
    hipStream_t kernel_stream;
    hipStreamCreate(&kernel_stream);
    infiniteKernel<<<grid, block, 0, kernel_stream>>>();
    hipEvent_t kernel_event;
    ErrorCheck(hipEventCreateWithFlags(&kernel_event, hipEventBlockingSync), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(kernel_event, kernel_stream),__FILE__, __LINE__);

    //wait for data copy to complete
    hipEventSynchronize(kernel_event);
    printf("Event kernel_event is finished\n");

    hipStreamDestroy(kernel_stream);
    hipEventDestroy(kernel_event);
    hipDeviceReset();

    return 0;
}
