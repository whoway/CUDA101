#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    printf("Matrix is: ");
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
    return;
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) C[i] = A[i] + B[i];
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 1 << 24;

    // malloc host pinned memory
    float *pinned_A, *pinned_B, *h_C;
    size_t nBytes = nElem * sizeof(float);
    ErrorCheck(hipHostAlloc((void**)&pinned_A, nBytes, hipHostMallocDefault), __FILE__, __LINE__);
    ErrorCheck(hipHostAlloc((void**)&pinned_B, nBytes, hipHostMallocDefault), __FILE__, __LINE__);
    h_C = (float*)malloc(nBytes);

    initialData(pinned_A, nElem);
    initialData(pinned_B, nElem);

    // allocate gpu global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipStream_t data_stream;
    hipStreamCreate(&data_stream);

    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);
    hipMemcpyAsync(d_B, pinned_B, nBytes, hipMemcpyHostToDevice, data_stream);
    hipStreamSynchronize(data_stream);



    //calculate on GPU
    dim3 block (512);
    dim3 grid  ((nElem + block.x - 1)/ block.x, 1);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);

    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

    for (int i = nElem - 1; i >  nElem - 50; i--)
    {
        printf("ElemIdx=%d, matrix_A:%.2f, matrix_B:%.2f, result=%.2f\n", i, pinned_A[i],pinned_B[i],h_C[i] );
    }
    hipHostFree(pinned_A);
    hipHostFree(pinned_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(data_stream);
    hipDeviceReset();

    return 0;
}
