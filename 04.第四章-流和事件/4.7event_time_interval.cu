#include <hip/hip_runtime.h>
#include "../common/common.h"
#include <stdio.h>

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
    return;
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    //get the supported priority on this device
    int lowPriority = 0;
    int highPriority = 0;
    hipDeviceGetStreamPriorityRange(&lowPriority, &highPriority);
    printf("Priority Range is from %d to %d\n", lowPriority, highPriority);
    int nElem = 1 << 24;

    // malloc host pinned memory
    float *pinned_A;
    size_t nBytes = nElem * sizeof(float);
    ErrorCheck(hipHostAlloc((void**)&pinned_A, nBytes, hipHostMallocDefault), __FILE__, __LINE__);
    initialData(pinned_A, nElem);

    // allocate gpu global memory
    float *d_A;
    hipMalloc((float**)&d_A, nBytes);

    // transfer data from host to device
    hipStream_t data_stream;
    hipStreamCreate(&data_stream);

    hipEvent_t begin_event;
    ErrorCheck(hipEventCreate(&begin_event), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(begin_event, data_stream),__FILE__, __LINE__);
    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);

    hipEvent_t end_event;
    ErrorCheck(hipEventCreate(&end_event), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(end_event, data_stream),__FILE__, __LINE__);
    hipEventSynchronize(end_event);
    float timeElapse = 0.0;
    hipEventElapsedTime(&timeElapse, begin_event, end_event);
    printf("time elapse for data copy from host to device is :%.2f(ms)\n", timeElapse);

    hipHostFree(pinned_A);
    hipStreamDestroy(data_stream);
    hipEventDestroy(begin_event);
    hipEventDestroy(end_event);
    hipDeviceReset();

    return 0;
}
