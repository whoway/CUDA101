#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

extern __shared__ int dynamic_array[];

__global__ void dynamic_shared_mem()
{
    dynamic_array[threadIdx.x] = threadIdx.x;
    printf("access dynamic_array in kernel, dynamic_array[%d]=%d\n", threadIdx.x, dynamic_array[threadIdx.x]);
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    //get current shared memory mode
    hipSharedMemConfig SharedMemConfig;
    ErrorCheck(hipDeviceGetSharedMemConfig(&SharedMemConfig), __FILE__, __LINE__);
    printf("current shared memory mode:%d\n", SharedMemConfig);
    if(hipSharedMemBankSizeEightByte != SharedMemConfig)
    {
        SharedMemConfig = hipSharedMemBankSizeEightByte;
        ErrorCheck(hipDeviceSetSharedMemConfig(SharedMemConfig),__FILE__, __LINE__);

    }
    else if(hipSharedMemBankSizeFourByte != SharedMemConfig){
        SharedMemConfig = hipSharedMemBankSizeFourByte;
        ErrorCheck(hipDeviceSetSharedMemConfig(SharedMemConfig),__FILE__, __LINE__);

    }
    printf("current shared memory mode,after modify:%d\n", SharedMemConfig);


    hipDeviceReset();

    return 0;
}
