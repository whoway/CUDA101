#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ float factor = 0;

__global__ void globalMemory()
{
    printf("device global memory:%.2f\n", factor);
    factor += 1.2;
}

int main(int argc, char **argv)
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != 0 || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != 0)
    {
        printf("fail to set GPU 0 for computing\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    dim3 block(1, 1);
    dim3 grid(1,1);
    float h_A = 3.6;
    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_A, sizeof(float), 0, hipMemcpyHostToDevice),__FILE__, __LINE__);
    globalMemory<<<grid, block>>>();
    hipDeviceSynchronize();
    ErrorCheck(hipMemcpyFromSymbol(&h_A, HIP_SYMBOL(factor), sizeof(float), 0, hipMemcpyDeviceToHost), __FILE__, __LINE__);
    printf("hipMemcpyFromSymbol result is:%.2f\n", h_A);

    //get global address
    float *pd_A;
    ErrorCheck(hipGetSymbolAddress((void**)&pd_A, factor), __FILE__, __LINE__);
    hipMemcpy(&h_A, pd_A, sizeof(float), hipMemcpyDeviceToHost);
    printf("hipGetSymbolAddress result is:%.2f\n", h_A);

    hipDeviceReset();
    return (0);
}
