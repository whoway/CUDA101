#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ float factor;

__global__ void constantMemory()
{
    printf("Get constant memory:%.2f\n", factor);
}

int main(int argc, char **argv)
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != 0 || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != 0)
    {
        printf("fail to set GPU 0 for computing\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    dim3 block(8, 1);
    dim3 grid(1,1);
    float h_factor = 2.3;
    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_factor, sizeof(float), 0, hipMemcpyHostToDevice), __FILE__, __LINE__);
    constantMemory<<<grid, block>>>();
    hipDeviceSynchronize();
    // reset device
    hipDeviceReset();
    return (0);
}
