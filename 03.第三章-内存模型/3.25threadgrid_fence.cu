#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

__device__ int g_shared = 0;

__global__ void thread_grid_fence()
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id == 0)
    {
        g_shared = 5.0;
    }
    __threadfence();
 
    printf("access local shared in thread_fence, g_shared=%d, blockIdx=%d, threadIdx=%d, threadId=%d\n",
            g_shared, blockIdx.x, threadIdx.x, id);
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    //calculate on GPU
    dim3 block (32);
    dim3 grid  (2);

    thread_grid_fence<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
