#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ float factor = 3.2;

__global__ void globalMemory(float *out)
{
    printf("device global memory:%.2f\n", factor);
    *out = factor;
}

int main(int argc, char **argv)
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != 0 || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != 0)
    {
        printf("fail to set GPU 0 for computing\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    dim3 block(1, 1);
    dim3 grid(1,1);
    float *d_A;
    float h_A;

    hipMalloc((void **)&d_A, sizeof(float));
    globalMemory<<<grid, block>>>(d_A);
    hipDeviceSynchronize();
    hipMemcpy(&h_A, d_A, sizeof(float), hipMemcpyDeviceToHost);
    printf("Host memory:%.2f\n", h_A);
    hipFree(d_A);
    hipDeviceReset();
    return (0);
}
