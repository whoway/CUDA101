#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../common/common.h"
#include <stdio.h>

__global__ void sumArraysOnGPU(int *A, int *B, int *C, const int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) C[i] = A[i] + B[i];
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    //check whether L1 cache is supported
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if(deviceProp.globalL1CacheSupported)
    {
        printf("Global L1 cache is supported, %d!\n", deviceProp.globalL1CacheSupported);
    }
    else
    {
        printf("Global L1 cache is not supported, %d!\n", deviceProp.globalL1CacheSupported);
    }
    

    // set up data size of vectors
    int nElem = 1 << 24;

    // malloc host memory
    size_t nBytes = nElem * sizeof(int);

    int *h_A, *h_B, *gpuRef;
    h_A     = (int *)malloc(nBytes);
    h_B     = (int *)malloc(nBytes);
    gpuRef  = (int *)malloc(nBytes);
    if(NULL != h_A && NULL != h_B && NULL != gpuRef)
    {
        printf("allocate memory successfully\n");
    }
    else
    {
        printf("fail to allocate memory\n");
        return -1;
    }
    // initialize data at host side
    for(int i = 0; i < nElem; i++)
    {
        h_A[i] = i;
        h_B[i] = i + 1;
    }
    memset(gpuRef,  0, nBytes);
    //allocate GPU memory
    int *d_A, *d_B, *d_C;
    ErrorCheck(hipMalloc((int**)&d_A, nBytes), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((int**)&d_B, nBytes), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((int**)&d_C, nBytes), __FILE__, __LINE__);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    //calculate on GPU
    dim3 block (1024);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    double start_Time = GetCPUSecond();
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    double iElaps = GetCPUSecond() - start_Time;

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    for (int i = nElem - 1; i >  nElem - 50; i--)
    {
        printf("ElemSize=%d, index=%d, matrix_A:%d, matrix_B:%d, result=%d\n", nElem, i+1, h_A[i],h_B[i],gpuRef[i] );
    }
    printf("sumArraysOnGPU <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
            grid.y, block.x, block.y, iElaps);
    free(h_A);
    free(h_B);
    free(gpuRef);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();

    return 0;
}
