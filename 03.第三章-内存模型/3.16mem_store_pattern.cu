#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    printf("Matrix is: ");
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
    return;
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N, const int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    if (k < N) C[k] = A[i] + B[i];
}


int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 1 << 12;

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);
    if(NULL != h_A && NULL != h_B && NULL != gpuRef)
    {
        printf("allocate memory successfully\n");
    }
    else
    {
        printf("fail to allocate memory\n");
        return -1;
    }
    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(gpuRef,  0, nBytes);
    //allocate GPU memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    if(d_A == NULL || d_B == NULL || d_C == NULL){
        printf("fail to allocate memory for GPU\n");
        free(h_A);
        free(h_B);
        free(gpuRef);
        return -1;
    }
    else
    {
        printf("successfully allocate memory for GPU\n");
    }

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    //calculate on GPU
    dim3 block (256);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);
    int offset = 3;
    if(2 == argc)
    {
        offset = atoi(argv[1]);
    }
    double dTime_Begin = GetCPUSecond();
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    hipDeviceSynchronize();
    double dTime_End = GetCPUSecond();
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);


    for (int i = nElem - 1; i > nElem - 50; i--)
    {
        printf("idx=%d, matrix_A:%.2f, matrix_B:%.2f, result=%.2f\n", i+1, h_A[i],h_B[i],gpuRef[i] );
    }

    printf("Element Size:%d, threadConfig:<<< %d, %d >>>, offset:%d, Matrix add time Elapse is:%.5f\n", nElem,     grid.x, block.x, offset, dTime_End - dTime_Begin);
    free(h_A);
    free(h_B);
    free(gpuRef);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();

    return 0;
}
