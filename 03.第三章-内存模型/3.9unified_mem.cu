#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>

__managed__ float y = 9.0;

__global__ void unifiedMemory(float *A)
{
    *A +=  y;
    printf("GPU unified memory:%.2f\n", *A);
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    //check whether to support unified memory
    int supportManagedMemory = 0;
    ErrorCheck(hipDeviceGetAttribute(&supportManagedMemory, hipDeviceAttributeManagedMemory, dev), __FILE__, __LINE__);

    if(0 == supportManagedMemory)
    {
        printf("allocate managed memory is not supported\n");
        return -1;
    }
    printf("unified memory model is supported:%d\n", supportManagedMemory);

    //calculate on GPU
    dim3 block (1);
    dim3 grid  (1);
    float *unified_mem = NULL;
    ErrorCheck(hipMallocManaged((void**)&unified_mem, sizeof(float), hipMemAttachGlobal),__FILE__, __LINE__);

    *unified_mem = y;
    unifiedMemory<<<grid, block>>>(unified_mem);
    hipDeviceSynchronize();
    printf("CPU unified memory:%.2f\n", *unified_mem);
    hipFree(unified_mem);
    hipDeviceReset();
    return 0;
}
