#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>


__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    values_read[tid] = atomicAdd(shared_var, 1);

    for (i = 0; i < iters; i++)
    {
        atomicAdd(shared_var, 1);
    }
}


__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    int old = *shared_var;
    *shared_var = old + 1;
    values_read[tid] = old;

    for (i = 0; i < iters; i++)
    {
        int old = *shared_var;
        *shared_var = old + 1;
    }
}

static void print_read_results(int *h_arr, int *d_arr, int N,
                               const char *label)
{
    int i;
    hipMemcpy(h_arr, d_arr, N * sizeof(int),
                     hipMemcpyDeviceToHost);
    printf("Threads performing %s operations read values", label);

    for (i = 0; i < N; i++)
    {
        printf(" %d", h_arr[i]);
    }

    printf("\n");
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    int N = 64;
    int block = 32;
    int runs = 30;
    int iters = 100000;
    int r;
    int *d_shared_var;
    int h_shared_var_atomic, h_shared_var_unsafe;
    int *d_values_read_atomic;
    int *d_values_read_unsafe;
    int *h_values_read;

    hipMalloc((void **)&d_shared_var, sizeof(int));
    hipMalloc((void **)&d_values_read_atomic, N * sizeof(int));
    hipMalloc((void **)&d_values_read_unsafe, N * sizeof(int));
    h_values_read = (int *)malloc(N * sizeof(int));

    double atomic_mean_time = 0;
    double unsafe_mean_time = 0;

    for (r = 0; r < runs; r++)
    {
        double start_atomic = GetCPUSecond();
        hipMemset(d_shared_var, 0x00, sizeof(int));
        atomics<<<N / block, block>>>(d_shared_var, d_values_read_atomic, N,
                                          iters);
        hipDeviceSynchronize();
        atomic_mean_time += GetCPUSecond() - start_atomic;
        hipMemcpy(&h_shared_var_atomic, d_shared_var, sizeof(int),
                         hipMemcpyDeviceToHost);

        double start_unsafe = GetCPUSecond();
        hipMemset(d_shared_var, 0x00, sizeof(int));
        unsafe<<<N / block, block>>>(d_shared_var, d_values_read_unsafe, N,
                                         iters);
        hipDeviceSynchronize();
        unsafe_mean_time += GetCPUSecond() - start_unsafe;
        hipMemcpy(&h_shared_var_unsafe, d_shared_var, sizeof(int),
                         hipMemcpyDeviceToHost);
    }

    printf("In total, %d runs using atomic operations took %f s\n",
           runs, atomic_mean_time);
    printf("  Using atomic operations also produced an output of %d\n",
           h_shared_var_atomic);
    printf("In total, %d runs using unsafe operations took %f s\n",
           runs, unsafe_mean_time);
    printf("  Using unsafe operations also produced an output of %d\n",
           h_shared_var_unsafe);

    print_read_results(h_values_read, d_values_read_atomic, N, "atomic");
    print_read_results(h_values_read, d_values_read_unsafe, N, "unsafe");
    hipFree(d_shared_var);
    hipFree(d_values_read_atomic);
    hipFree(d_values_read_unsafe);
    free(h_values_read);
    hipDeviceReset();

    return 0;
}
