#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <stdlib.h>


__global__ void fmad_kernel(double x, double y, double *out)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid == 0)
    {
        *out = x * x + y;
    }
}

double host_fmad_kernel(double x, double y)
{
    return x * x + y;
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    double *d_out, h_out;
    double x = 2.891903;
    double y = -3.980364;

    double host_value = host_fmad_kernel(x, y);
    hipMalloc((void **)&d_out, sizeof(double));
    fmad_kernel<<<1, 32>>>(x, y, d_out);
    hipMemcpy(&h_out, d_out, sizeof(double), hipMemcpyDeviceToHost);

    if (host_value == h_out)
    {
        printf("The device output the same value as the host.\n");
    }
    else
    {
        printf("The device output a different value than the host, diff=%e.\n",
               fabs(host_value - h_out));
    }

    return 0;
}
