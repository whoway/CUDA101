#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>


#define N   2
#define M   2

__device__ int foo(int row, int col)
{
    return (2 * row);
}

__global__ void kernel(int **arr)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;

    for ( ; tid < N; tid++)
    {
        for (i = 0; i < M; i++)
        {
            arr[5][i] = foo(tid, i);
        }
    }
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    int i;

    int **h_matrix;

    int **d_matrix;

    h_matrix = (int **)malloc(N * sizeof(int *));

    hipMalloc((void **)&d_matrix, N * sizeof(int *));
    hipMemset(d_matrix, 0x00, N * sizeof(int *));

    int **d_ptrs;
    d_ptrs = (int **)malloc(N * sizeof(int *));

    for (i = 0; i < N; i++)
    {
        h_matrix[i] = (int *)malloc(M * sizeof(int));
        hipMalloc((void **)&(d_ptrs[i]), M * sizeof(int));
        hipMemset(d_ptrs[i], 0x00, M * sizeof(int));
    }
    hipMemcpy(d_matrix, d_ptrs, N * sizeof(int *),hipMemcpyHostToDevice);

    int threadsPerBlock = 2;
    int blocksPerGrid = 2;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_matrix);

    // Copy rows back
    for (i = 0; i < N; i++)
    {
        hipMemcpy(h_matrix[i], d_matrix[i], M * sizeof(int),hipMemcpyDeviceToHost);
        hipFree(d_matrix[i]);
        free(h_matrix[i]);
    }

    hipFree(d_matrix);
    free(h_matrix);

    return 0;
}
