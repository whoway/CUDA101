#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>


#define N   2
#define M   2

__device__ int foo(int row, int col)
{
    return (2 * row);
}

__global__ void kernel(int **arr)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;

    for ( ; tid < N; tid++)
    {
        for (i = 0; i < M; i++)
        {
            arr[tid][i] = foo(tid, i);
        }
    }
}

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    int i;

    int **h_matrix;
    int **d_ptrs;

    h_matrix = (int **)malloc(N * sizeof(int *));
    d_ptrs = (int **)malloc(N * sizeof(int *));

    int **d_matrix;
    hipMalloc((void **)&d_matrix, N * sizeof(int *));
    hipMemset(d_matrix, 0x00, N * sizeof(int *));

    for (i = 0; i < N; i++)
    {
        h_matrix[i] = (int *)malloc(M * sizeof(int));
        hipMalloc((void **)&(d_ptrs[i]), M * sizeof(int));
        hipMemset(d_ptrs[i], 0x00, M * sizeof(int));
    }

    int threadsPerBlock = 2;
    int blocksPerGrid = 2;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_matrix);

    // Copy rows back
    for (i = 0; i < N; i++)
    {
        hipMemcpy(h_matrix[i], d_ptrs[i], M * sizeof(int),hipMemcpyDeviceToHost);
        hipFree(d_ptrs[i]);
        free(h_matrix[i]);
    }

    hipFree(d_matrix);
    free(h_matrix);

    return 0;
}
