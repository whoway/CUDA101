#include <hip/hip_runtime.h>
#include "../common/common.h"
#include <stdio.h>

int main(int argc, char **argv)
{
    //get GPU decice count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return -1;
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    //allocate GPU memory
    float *d_A;
    hipMalloc((float**)&d_A, sizeof(float) * 10);
    hipFree(d_A);
    hipFree(d_A);
    hipDeviceReset();

    return 0;
}
