#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mathKernel(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if (((tid / 32) % 2) == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

double GetCPUSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char **argv)
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != 0 || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != 0)
    {
        printf("fail to set GPU 0 for computing\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    // set up data size
    int size = 64;
    int blocksize = 64;

    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    error = ErrorCheck(hipMalloc((float**)&d_C, nBytes), __FILE__, __LINE__);
    if(error != 0)
    {
        printf("fail to allocate memory for GPU\n");
        return -1;
    }

    double iStart = GetCPUSecond();
    mathKernel<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    double iElaps = GetCPUSecond() - iStart;
    printf("mathKernel <<< %d %d >>> elapsed %.4f sec \n", grid.x, block.x,iElaps );

    // free gpu memory and reset divece
    hipFree(d_C);
    hipDeviceReset();
    return 0;
}
