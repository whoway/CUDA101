#include "common/common.h"
#include <stdio.h>

int main(int argc, char **argv)
{
    float* gpuMemory = NULL;
    ErrorCheck(hipMalloc(&gpuMemory, sizeof(float)), __FILE__, __LINE__);
    ErrorCheck(hipFree(gpuMemory), __FILE__, __LINE__);
    ErrorCheck(hipFree(gpuMemory), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);

    return 1;
}


