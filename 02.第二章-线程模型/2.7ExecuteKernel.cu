#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void helloFromGPU()
{
    printf("Hello World from GPU\n", );

}

int main(int argc, char **argv)
{
    printf("Hello World from CPU\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}


