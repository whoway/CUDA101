#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

template <unsigned int iBlockSize>
__global__ void reduceCompleteUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    // unrolling 8
    if(idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2* blockDim.x];
        int a4 = g_idata[idx + 3* blockDim.x];
        int b1 = g_idata[idx + 4* blockDim.x];
        int b2 = g_idata[idx + 5* blockDim.x];
        int b3 = g_idata[idx + 6* blockDim.x];
        int b4 = g_idata[idx + 7* blockDim.x];
        g_idata[idx] = a1 + a2 + a3+ a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();
    // in-place reduction and complete unroll
    if (iBlockSize>=1024 && tid < 512) idata[tid] += idata[tid + 512];
        __syncthreads();
    if (iBlockSize>=512 && tid < 256) idata[tid] += idata[tid + 256];
        __syncthreads();
    if (iBlockSize>=256 && tid < 128) idata[tid] += idata[tid + 128];
        __syncthreads();
    if (iBlockSize>=128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();
    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


int main(int argc, char **argv)
{
    int blocksize = atoi(argv[1]);
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        return (-1);
    }
    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // initialization
    int size = 1 << 24; // total number of elements
    printf("    with array size %d  ", size);

    
    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = (int)( rand() & 0xFF );
    }
    memcpy (tmp, h_idata, bytes);
    double iStart, iElaps;
    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x * sizeof(int));

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    iStart = GetCPUSecond();
    switch (blocksize)
    {
        case 1024:
            reduceCompleteUnroll<1024><<<grid.x/8, block>>>(d_idata, d_odata, size);
            break;
        case 512:
            reduceCompleteUnroll<512><<<grid.x/8, block>>>(d_idata, d_odata, size);
            break;
        case 256:
            reduceCompleteUnroll<256><<<grid.x/8, block>>>(d_idata, d_odata, size);
            break;
        case 128:
            reduceCompleteUnroll<128><<<grid.x/8, block>>>(d_idata, d_odata, size);
            break;
        case 64:
            reduceCompleteUnroll<64><<<grid.x/8, block>>>(d_idata, d_odata, size);
            break;
    }
    hipDeviceSynchronize();
    iElaps = GetCPUSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);
    free(tmp);

    // free device memory
    hipFree(d_idata);
    hipFree(d_odata);

    // reset device
    hipDeviceReset();
    return 0;
}
